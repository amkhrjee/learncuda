
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void deviceInfo(int *device)
{
    hipGetDevice(device);
}

int main(void)
{
    int count;
    int *device;
    int h_device;

    hipGetDeviceCount(&count);

    hipMalloc(&device, sizeof(int));

    std::cout << "Device count: " << count << std::endl;

    deviceInfo<<<1, 1>>>(device);

    hipMemcpy(&h_device, device, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Device: " << h_device << std::endl;

    return 0;
}