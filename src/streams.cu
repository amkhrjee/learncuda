#include "hip/hip_runtime.h"
#include "cuda_error.h"
#include <iostream>
#include <iomanip>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void)
{
    hipDeviceProp_t props;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(cudaGetDeviceProperties_v2(&props, whichDevice));
    if (!props.deviceOverlap)
    {
        std::cout << "Device does not support streams :(" << std::endl;
        exit(EXIT_FAILURE);
    }

    int *host_a, *host_b, *host_c;

    // allocate page-locked memory used to stream
    HANDLE_ERROR(hipHostAlloc(&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc(&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc(&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // initialiazing the stream
    hipStream_t stream0, stream1;
    HANDLE_ERROR(hipStreamCreate(&stream0));
    HANDLE_ERROR(hipStreamCreate(&stream1));

    int *dev_a0, *dev_b0, *dev_c0; // For stream0
    int *dev_a1, *dev_b1, *dev_c1; // For stream1

    HANDLE_ERROR(hipMalloc(&dev_a0, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_b0, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_c0, N * sizeof(int)));

    HANDLE_ERROR(hipMalloc(&dev_a1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_b1, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_c1, N * sizeof(int)));

    hipEvent_t start, stop;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // Launch kernels in chunks of N
    for (int i = 0; i < FULL_DATA_SIZE; i += N * 2)
    {
        // For stream0
        HANDLE_ERROR(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
        HANDLE_ERROR(hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        HANDLE_ERROR(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
        HANDLE_ERROR(hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
        // For stream1

        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

        // Copy the data from device to locked memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0));
        HANDLE_ERROR(hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));
    }

    HANDLE_ERROR(hipStreamSynchronize(stream0));
    HANDLE_ERROR(hipStreamSynchronize(stream1));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Time taken:" << elapsedTime << "ms" << std::endl;
    // cleanup the streams and memory
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a0));
    HANDLE_ERROR(hipFree(dev_b0));
    HANDLE_ERROR(hipFree(dev_c0));
    HANDLE_ERROR(hipFree(dev_a1));
    HANDLE_ERROR(hipFree(dev_b1));
    HANDLE_ERROR(hipFree(dev_c1));

    HANDLE_ERROR(hipStreamDestroy(stream0));
    HANDLE_ERROR(hipStreamDestroy(stream1));

    return 0;
}