#include "hip/hip_runtime.h"
#include "cuda_error.h"
#include <iostream>
#include <iomanip>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void)
{
    hipDeviceProp_t props;
    int whichDevice;
    HANDLE_ERROR(hipGetDevice(&whichDevice));
    HANDLE_ERROR(cudaGetDeviceProperties_v2(&props, whichDevice));
    if (!props.deviceOverlap)
        std::cout << "Device does not support streams :(" << std::endl;

    hipEvent_t start, stop;
    float elapsedTime;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc(&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_c, N * sizeof(int)));

    // allocate page-locked memory used to stream
    HANDLE_ERROR(hipHostAlloc(&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc(&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc(&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // initialiazing the stream
    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));

    // Launch kernels in chunks of N
    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
        HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        // Copy the data from device to locked memory
        HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));

        // Make sure all ops on the stream are finished
        HANDLE_ERROR(hipStreamSynchronize(stream));
    }

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Time taken:" << elapsedTime << "ms" << std::endl;
    // cleanup the streams and memory
    HANDLE_ERROR(hipHostFree(host_a));
    HANDLE_ERROR(hipHostFree(host_b));
    HANDLE_ERROR(hipHostFree(host_c));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));

    HANDLE_ERROR(hipStreamDestroy(stream));

    return 0;
}