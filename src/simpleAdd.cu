
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main(void)
{
    int c;
    int *dev_c;

    hipError_t err = hipMalloc(&dev_c, sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    add<<<1, 1>>>(2, 7, dev_c);

    err = hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("2 + 7 = %d\n", c);

    hipFree(dev_c);

    return 0;
}