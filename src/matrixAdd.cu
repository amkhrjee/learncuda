
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10

__global__ void MatAdd(int A[N][N], int B[N][N], int C[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

int main(void)
{
    int A[N][N], B[N][N], C[N][N];
    int (*d_A)[N], (*d_B)[N], (*d_C)[N];

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i][j] = i * j;
            B[i][j] = i + j;
        }
    }

    hipMalloc(&d_A, sizeof(int) * N * N);
    hipMalloc(&d_B, sizeof(int) * N * N);
    hipMalloc(&d_C, sizeof(int) * N * N);

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    int numBlocks = 1;
    //  We're running one 2D thread block of dim N x N
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%d\t", C[i][j]);
        printf("\n");
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}