#include "hip/hip_runtime.h"
#include "cuda_error.h"
#include <iostream>
#include <iomanip>

#define SIZE 100 * 1024 * 1024

void *big_random_block(int size)
{
    unsigned char *data = (unsigned char *)malloc(size);
    HANDLE_NULL(data);
    for (int i = 0; i < size; i++)
        data[i] = rand();

    return data;
}

__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo)
{
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0; // this initializes all indices as 0
    __syncthreads();
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < size)
    {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }

    __syncthreads();
    atomicAdd(&histo[threadIdx.x], temp[threadIdx.x]);
}

int main(void)
{
    unsigned char *buffer = (unsigned char *)big_random_block(SIZE);
    unsigned int histo[256];

    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    unsigned char *dev_buffer;
    unsigned int *dev_histo;

    HANDLE_ERROR(hipMalloc(&dev_buffer, SIZE));
    HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMalloc(&dev_histo, 256 * sizeof(long)));
    HANDLE_ERROR(hipMemset(dev_histo, 0, 256 * sizeof(int)));

    hipDeviceProp_t props;
    HANDLE_ERROR(cudaGetDeviceProperties_v2(&props, 0));
    int procs = props.multiProcessorCount; // Empirical result
    histo_kernel<<<2 * procs, 256>>>(dev_buffer, SIZE, dev_histo);

    HANDLE_ERROR(hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elasped_time;
    HANDLE_ERROR(hipEventElapsedTime(&elasped_time, start, stop));

    std::cout << std::fixed << std::setprecision(3);
    std::cout << "Elapsed time: " << elasped_time << "ms" << std::endl;

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    free(buffer);
    hipFree(dev_buffer);
    hipFree(dev_histo);
    return 0;
}